
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define m 2048
#define n 2048
//#define size 5

#define tile_width 16

using namespace std;

/*
__global__ void add_1(int a[], int b[], int c[], int fila, int columna)
{
        int index =(blockDim.x * blockIdx.x)+threadIdx.x;
        if(blockIdx.x<fila && threadIdx.x<columna)
                c[index]=a[index]+b[index];
}
*/

__global__ void mult_matrix(int a[],int b[], int c[],int fila,int columna)
{
        int row = blockIdx.y*blockDim.y+threadIdx.y;
        int col = blockIdx.x*blockDim.x+threadIdx.x;

        if((row<fila)&&(col<columna))
        {
                int Pvalue = 0;
                for(int k=0;k<fila;++k)
                {
                        Pvalue += a[row*fila+k]*b[k*columna+col];
                }
                c[row*fila+col] = Pvalue;
        }
}


__global__ void mult_matrix_tile(int a[], int b[], int c[],int fila,int columna){
        __shared__ int a_ds[tile_width][tile_width];
        __shared__ int b_ds[tile_width][tile_width];

        int bx = blockIdx.x; int by=blockIdx.y;
        int tx = threadIdx.x; int ty=threadIdx.y;

        int row = by * tile_width + ty;
        int col = bx * tile_width + tx;

        float Pvalue = 0;
        for(int ph=0;ph<fila/tile_width;++ph)
        {
                a_ds[ty][tx] = a[row*fila+ ph*tile_width + tx];
                b_ds[ty][tx] = b[(ph*tile_width+ty)*fila+col];
                __syncthreads();

                for(int k=0;k<tile_width;++k)
                {
                        Pvalue += a_ds[ty][k]*b_ds[k][tx];
                }
                __syncthreads();
        }
        c[row*fila+col] = Pvalue;
}

__global__ void mult_matrix_tile_2(int a[], int b[], int c[],int fila,int columna){
        __shared__ int a_ds[tile_width][tile_width];
        __shared__ int b_ds_0[tile_width][tile_width];
        __shared__ int b_ds_1[tile_width][tile_width];
        __shared__ int b_ds_2[tile_width][tile_width];
        __shared__ int b_ds_3[tile_width][tile_width];

        const uint bx = blockIdx.x; 
        const uint by=blockIdx.y;
        
        const uint tx = threadIdx.x; 
        const uint ty=threadIdx.y;

        const uint row = by * tile_width + ty;

        const uint col_0 = (4*bx+0 )* tile_width + tx;
        const uint col_1 = (4*bx+1 )* tile_width + tx;
        const uint col_2 = (4*bx+2 )* tile_width + tx;
        const uint col_3 = (4*bx+3 )* tile_width + tx;


        float Pvalue_0 = 0.0f, Pvalue_1 = 0.0f, Pvalue_2 = 0.0f, Pvalue_3 = 0.0f; 
        for(uint ph=0;ph<fila/tile_width;++ph)
        {
                a_ds[ty][tx] = a[row*fila+ ph*tile_width + tx];

                b_ds_0[ty][tx] = b[(ph*tile_width+ty)*fila+col_0];
                b_ds_1[ty][tx] = b[(ph*tile_width+ty)*fila+col_1];
                b_ds_2[ty][tx] = b[(ph*tile_width+ty)*fila+col_2];
                b_ds_3[ty][tx] = b[(ph*tile_width+ty)*fila+col_3];
                
                __syncthreads();

                for(uint k=0;k<tile_width;++k)
                {
                        Pvalue_0 += a_ds[ty][k]*b_ds_0[k][tx];
                        Pvalue_1 += a_ds[ty][k]*b_ds_1[k][tx];
                        Pvalue_2 += a_ds[ty][k]*b_ds_2[k][tx];
                        Pvalue_3 += a_ds[ty][k]*b_ds_3[k][tx];
                }
                __syncthreads();
        }
        c[row*fila+col_0] = Pvalue_0;
        c[row*fila+col_1] = Pvalue_1;
        c[row*fila+col_2] = Pvalue_2;
        c[row*fila+col_3] = Pvalue_3;
}

void llenar_random_matrix(int a[],int f ,int c)
{
        for(int i=0; i<f ;i++)
        {
                for(int j=0; j<c;j++)
                {
                        a[(i*n)+j] = rand()%3;
                }
        }
}

void print_matrix(int a[],int f,int c)
{
        for(int i=0; i<f;i++)
        {
                for(int j=0; j<c;j++)
                        printf("%d ",a[(i*n)+j]);
                printf("\n");
        }
}


int main(int argc, char * argv[]){

        int *a, *b, *c;
        int *d_a, *d_b, *d_c;


        int size = m*n*sizeof(int);

        //separar espacion de memoria para copias en device

        float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

        // some events to count the execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // start to count execution time of GPU version
        hipEventRecord(start, 0);

        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        //separa espacio en host

        a = (int *)malloc(size);
        llenar_random_matrix(a,m,n);
        b = (int *)malloc(size);
        llenar_random_matrix(b,m,n);
        c = (int *)malloc(size);
        // imprimir matrices a , b

        print_matrix(a, m, n);
        printf("-----------------\n");
        print_matrix(b, m, n);
        printf("-----------------\n");
        // copias entrada a device

        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // lanzar funcion kernel en gpu con N blocks

       
        unsigned int grid_rows = (m)/tile_width;
        unsigned int grid_cols = (n)/tile_width;

        //printf("m+tile -1: %d\n",m+tile_width-1);
        printf("g_row: %d\n",grid_rows);
        printf("g_col: %d\n",grid_cols);

        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(tile_width,tile_width);
        mult_matrix_tile<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,m,n);
        //mult_matrix<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,m,n);
        // add_3<<<n,1>>>(d_a,d_b,d_c,m,n);

        // copia resultado al host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        //imprimir c result

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // compute time elapse on GPU computing
        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
        printf("Time elapsed on matrix multiplication of GPU: %f : \n",  gpu_elapsed_time_ms);

        print_matrix(c,m,n);
        // limpiar memoria
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b);hipFree(d_c);
        return 0;
}

//// tiled Time elapsed on matrix multiplication of GPU: 1.258784 :

/// normal 
