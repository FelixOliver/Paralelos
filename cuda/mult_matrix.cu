
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define m 2048
#define n 2048
//#define size 5

#define tile_width 16

using namespace std;

/*
__global__ void add_1(int a[], int b[], int c[], int fila, int columna)
{
        int index =(blockDim.x * blockIdx.x)+threadIdx.x;
        if(blockIdx.x<fila && threadIdx.x<columna)
                c[index]=a[index]+b[index];
}
*/

__global__ void mult_matrix(int a[],int b[], int c[],int fila,int columna)
{
        int row = blockIdx.y*blockDim.y+threadIdx.y;
        int col = blockIdx.x*blockDim.x+threadIdx.x;

        if((row<fila)&&(col<columna))
        {
                int Pvalue = 0;
                for(int k=0;k<fila;++k)
                {
                        Pvalue += a[row*fila+k]*b[k*columna+col];
                }
                c[row*fila+col] = Pvalue;
        }
}

__global__ void mult_matrix_tile(int a[], int b[], int c[],int fila,int columna){
        __shared__ int a_ds[tile_width][tile_width];
        __shared__ int b_ds[tile_width][tile_width];

        int bx = blockIdx.x; int by=blockIdx.y;
        int tx = threadIdx.x; int ty=threadIdx.y;

        int row = by * tile_width + ty;
        int col = bx * tile_width + tx;

        float Pvalue = 0;
        for(int ph=0;ph<fila/tile_width;++ph)
        {
                a_ds[ty][tx] = a[row*fila+ ph*tile_width + tx];
                b_ds[ty][tx] = b[(ph*tile_width+ty)*fila+col];
                __syncthreads();

                for(int k=0;k<tile_width;++k)
                {
                        Pvalue += a_ds[ty][k]*b_ds[k][tx];
                }
                __syncthreads();
        }
        c[row*fila+col] = Pvalue;
}

void llenar_random_matrix(int a[],int f ,int c)
{
        for(int i=0; i<f ;i++)
        {
                for(int j=0; j<c;j++)
                {
                        a[(i*n)+j] = rand()%3;
                }
        }
}

void print_matrix(int a[],int f,int c)
{
        for(int i=0; i<f;i++)
        {
                for(int j=0; j<c;j++)
                        printf("%d ",a[(i*n)+j]);
                printf("\n");
        }
}


int main(int argc, char * argv[]){

        int *a, *b, *c;
        int *d_a, *d_b, *d_c;


        int size = m*n*sizeof(int);

        //separar espacion de memoria para copias en device

        float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

        // some events to count the execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // start to count execution time of GPU version
        hipEventRecord(start, 0);

        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        //separa espacio en host

        a = (int *)malloc(size);
        llenar_random_matrix(a,m,n);
        b = (int *)malloc(size);
        llenar_random_matrix(b,m,n);
        c = (int *)malloc(size);
        // imprimir matrices a , b

        print_matrix(a, m, n);
        printf("-----------------\n");
        print_matrix(b, m, n);
        printf("-----------------\n");
        // copias entrada a device

        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // lanzar funcion kernel en gpu con N blocks

       
        unsigned int grid_rows = (m)/tile_width;
        unsigned int grid_cols = (n)/tile_width;

        //printf("m+tile -1: %d\n",m+tile_width-1);
        printf("g_row: %d\n",grid_rows);
        printf("g_col: %d\n",grid_cols);

        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(tile_width,tile_width);
        mult_matrix_tile<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,m,n);
        //mult_matrix<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,m,n);
        // add_3<<<n,1>>>(d_a,d_b,d_c,m,n);

        // copia resultado al host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        //imprimir c result

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // compute time elapse on GPU computing
        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
        printf("Time elapsed on matrix multiplication of GPU: %f : \n",  gpu_elapsed_time_ms);

        print_matrix(c,m,n);
        // limpiar memoria
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b);hipFree(d_c);
        return 0;
}

//// tiled Time elapsed on matrix multiplication of GPU: 1.258784 :

/// normal 
