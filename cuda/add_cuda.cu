#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define m 3
#define n 4

using namespace std;

//empezando cuda
///example hello in cuda
/*
__global__ void mykernel(void){

}

float main(void){
        mykernel<<<1,1>>>();
        printf("HOLITAS \n");
        return 0;
}
*/
/*
__global__ void add_kernel(float *a, float *b, float *c){

        c[blockIdx.x] = a[blockIdx.x]+b[blockIdx.x];
}
*/

__global__ void add_1(float a[], float b[], float c[], float fila, float columna)
{
        float index =(blockDim.x * blockIdx.x)+threadIdx.x;
        if(blockIdx.x<fila && threadIdx.x<columna)
                c[index]=a[index]+b[index];
}

__global__ void add_2(float a[], float b[], float c[], float fila, float columna)
{
        float index = blockIdx.x;
        if(index<fila)
        {
                for(float i=0;i<columna; i++)
                {
                        c[index+i]= a[index+i]+b[index+i];
                }
        }
}

__global__ void add_3(float a[], float b[], float c[], float fila, float columna)
{
        float index = blockIdx.x;
        if(index<columna)
        {
                for(float i=0; i<fila; i++)
                {
                        c[index+(i*columna)] = a[index+(i*columna)]+b[index+(i*columna)];
                }
        }
}

void llenar_random_matrix(float a[],float f, float c)
{
        for(float i=0; i<f ;i++)
        {
                for(float j=0; j<c;j++)
                {
                        a[(i*n)+j] = rand()%50;
                }
        }
}


void print_matrix(float a[],float f,float c)
{
        for(float i=0; i<f;i++)
        {
                for(float j=0; j<c;j++)
                        printf("%d ",a[(i*n)+j]);
                printf("\n");
        }
}

int main(float argc, char * argv[]){

        float *a, *b, *c;
        float *d_a, *d_b, *d_c;
        float size = m*n*sizeof(float);

        //separar espacion de memoria para copias en device

        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        //separa espacio en host

        a = (float *)malloc(size);
        llenar_random_matrix(a,m,n);
        b = (float *)malloc(size);
        llenar_random_matrix(b,m,n);
        c = (float *)malloc(size);
        // imprimir matrices a , b

        print_matrix(a, m, n);
        printf("-----------------\n");
        print_matrix(b, m, n);
        printf("-----------------\n");
        // copias entrada a device

        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


        // lanzar funcion kernel en gpu con N blocks

        //add_1<<<m,n>>>(d_a,d_b,d_c,m,n);
        //add_2<<<m,1>>>(d_a,d_b,d_c,m,n);
        add_3<<<n,1>>>(d_a,d_b,d_c,m,n);

        // copia resultado al host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        //imprimir c result
        print_matrix(c,m,n);
        // limpiar memoria
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b);hipFree(d_c);
        return 0;
}

